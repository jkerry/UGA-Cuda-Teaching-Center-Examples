#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "PartialSumExample.h"

/**
 * This boolean value defined if debug information is printed within macros
 */
#define DEBUG true
/**
 * The number of items in the partial sum array
 */
#define ARR_SIZE 512



/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
/**
 * This macro changes the active device to the device with the provided integer index.
 */
#define SET_DEVICE(value) {																\
	hipDeviceProp_t devProp;																\
	hipGetDeviceProperties(&devProp, value);											\
	if(DEBUG)printf("Changing the gpu to device id: %i name: %s\n",value,devProp.name);	\
	CUDA_CHECK_RETURN(hipSetDevice(value));											\
																						\
}




__global__ void sumReduction(int* data, int length, int* dev_result){
	const unsigned int tid = threadIdx.x;
	//allocate shared memory within the block for the partial sums
	__shared__ int partialSum[ARR_SIZE];
	if(tid < length){
		partialSum[tid]=data[tid];
		//half the number of active threads at any time, compute the current partial sum
		for( unsigned int stride = 1; stride < blockDim.x; stride *= 2){
			__syncthreads();
			if(tid%(2*stride)==0){

				partialSum[tid]+=partialSum[tid+stride];
			}
		}
	}
	__syncthreads();
	//return the last computed partial sum
	if(tid==0){
		*dev_result = partialSum[0];
	}
	return;
}

__global__ void improved_sumReduction(int* data, int length, int* dev_result){
	const unsigned int tid = threadIdx.x;
	//allocate shared memory within the block for the partial sums
	__shared__ int partialSum[ARR_SIZE];
		if(tid < length){
			partialSum[tid]=data[tid];
			//half the number of active threads at any time, compute the current partial sum
			for( unsigned int stride = blockDim.x>>1; stride >0 ; stride >>=1){
				__syncthreads();
				if(tid < stride){
					partialSum[tid]+=partialSum[tid+stride];
				}
			}
			//return the last computed partial sum

		}
		__syncthreads();
		//return the last computed partial sum
		if(tid==0){
			*dev_result = partialSum[0];
		}
		return;
}






/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int partialSum(bool improved) {
	SET_DEVICE(0);
	CUDA_CHECK_RETURN(hipDeviceReset()); //pre-clear the device

	int *randomData, *randomData_Device, *dev_result;
	//generate random data
	randomData =  (int*) malloc(ARR_SIZE*sizeof(int));
	srand(time(NULL));
	for(int i = 0; i < ARR_SIZE; i++){
		randomData[i] = (int)(((float)rand()/RAND_MAX)*100);
	}
	//initialize device pointers

	//allocate arrData pointer on device
	CUDA_CHECK_RETURN(hipMalloc((void**) &randomData_Device, ARR_SIZE*sizeof(int)));
	//copy local generated data to the device
	CUDA_CHECK_RETURN(hipMemcpy((void*)randomData_Device, randomData, ARR_SIZE*sizeof(int), hipMemcpyHostToDevice));
	//allocate the array of size 1 for return value
	CUDA_CHECK_RETURN(hipMalloc((void**) &dev_result, sizeof(int)));


	//launch the kernel
	if(!improved){
		printf("Running naive sum reduction\n");
		sumReduction<<<dim3(1,1,1),dim3(ARR_SIZE,1,1),0,0>>>(randomData_Device, ARR_SIZE, dev_result);
	}
	else{
		printf("Running improved sum reduction\n");
		improved_sumReduction<<<dim3(1,1,1),dim3(ARR_SIZE,1,1),0,0>>>(randomData_Device, ARR_SIZE, dev_result);
	}
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	int *GPU_Answer, Serial_Answer=0;
	GPU_Answer = (int*)malloc(sizeof(int));

	CUDA_CHECK_RETURN(hipMemcpy(GPU_Answer, dev_result, sizeof(int), hipMemcpyDeviceToHost));
	//compute the serial answer

	for(int i = 0; i < ARR_SIZE; i++){
		Serial_Answer += randomData[i];
	}
	printf("GPU Answer:\t%i\nSerial Answer:\t%i\n",*GPU_Answer,Serial_Answer);
	//Clean up
	free(GPU_Answer);
	free(randomData);
	hipFree(randomData_Device);
	CUDA_CHECK_RETURN(hipDeviceReset()); //clear the device after all work is completed
	return 0;
}
