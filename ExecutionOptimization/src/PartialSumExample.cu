#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "PartialSumExample.h"

/**
 * This boolean value defined if debug information is printed within macros
 */
#define DEBUG true
/**
 * The number of items in the partial sum array
 */
#define ARR_SIZE 512



/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
/**
 * This macro changes the active device to the device with the provided integer index.
 */
#define SET_DEVICE(value) {																\
	hipDeviceProp_t devProp;																\
	hipGetDeviceProperties(&devProp, value);											\
	if(DEBUG)printf("Changing the gpu to device id: %i name: %s\n",value,devProp.name);	\
	CUDA_CHECK_RETURN(hipSetDevice(value));											\
																						\
}


__device__ int result;
__global__ void sumReduction(int* data, int length){
	const unsigned int tid = threadIdx.x;
	if(tid < length){
		//allocate shared memory within the block for the partial sums
		__shared__ float partialSum[ARR_SIZE];
		//half the number of active threads at any time, compute the current partial sum
		for( unsigned int stride = 1; stride < blockDim.x; stride *= 2){
			__syncthreads();
			if(tid%(2*stride==0)){
				partialSum[tid]+=partialSum[tid+stride];
			}
		}
		//return the last computed partial sum
		result = partialSum[0];
		return;
	}
	else{ 	//these threads are idle, thread id is outside the array bounds
		return;
	}
}

/**
 * Wrapper function to initialize host data elements so that the example appears simpler.
 */
void ps_setupData(int* arrData, int arrData_length){
	arrData =  (int*) malloc(ARR_SIZE*sizeof(int));
	srand(time(NULL));
	for(int i = 0; i < arrData_length; i++){
		arrData[i] = rand();
	}
}
/**
 * wrapper function to initialize device element pointers so that the example appears simpler
 */
void ps_setupDeviceData(int* arrData, int arrData_length, int* arrData_Local,int* psResult){
	//allocate arrData pointer on device
	CUDA_CHECK_RETURN(hipMalloc((void**) &arrData, arrData_length*sizeof(int)));
	//copy local generated data to the device
	CUDA_CHECK_RETURN(hipMemcpy((void*)arrData, arrData_Local, arrData_length*sizeof(int), hipMemcpyHostToDevice));
	//allocate the array of size 1 for return value
	CUDA_CHECK_RETURN(hipMalloc((void**) &psResult, 1*sizeof(int)));
	//CUDA_CHECK_RETURN();
}



/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int partialSum(void) {
	int *randomData, *randomData_Device, *psResult_Device;
	ps_setupData(randomData, ARR_SIZE);
	ps_setupDeviceData(randomData_Device, ARR_SIZE,randomData, psResult_Device);

	SET_DEVICE(0);
	CUDA_CHECK_RETURN(hipDeviceReset()); //pre-clear the device
	//launch the kernel
	sumReduction<<<dim3(1,1,1),dim3(ARR_SIZE,1,1),0,0>>>(randomData_Device, ARR_SIZE);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
//	CUDA_CHECK_RETURN();
	CUDA_CHECK_RETURN(hipDeviceReset()); //clear the device after all work is completed
	return 0;
}
