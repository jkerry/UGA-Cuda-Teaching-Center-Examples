
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "PartialSumExample.h"



/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {

	partialSum(false);
	printf("Finished with naive sum reduction\n");
	partialSum(true);
	printf("Finished with improved sum reduction\n");

	return 0;
}
